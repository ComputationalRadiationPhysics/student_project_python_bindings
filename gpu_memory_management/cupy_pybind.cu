#include <stdio.h>
#include <string>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>


//store C++ and CUDA version of phase retrieval
#include "gpu_algo.hpp"

namespace py = pybind11;
using namespace std;

PYBIND11_MODULE(gpuMemManagement, m) 
{
  //cuda
  m.def("getDeviceNumber", &getDeviceNumber);
  m.def("getNumberofSM", &getNumberofSM);

  m.def("copy_to_device", &copy_to_device);
  m.def("update_images", &update_images);
  m.def("free_gpu_memory", &free_gpu_memory);

  //cuda stream test
  m.def("update_images_stream", &update_images_stream);

  
}



