#include <stdio.h>
#include <string>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>
#include <pybind11/stl.h>


//store C++ and CUDA version of phase retrieval
#include "gpu_algo.hpp"

namespace py = pybind11;
using namespace std;

PYBIND11_MODULE(gpuMemManagement, m) 
{
  //cuda
  m.def("getDeviceNumber", &getDeviceNumber);
  m.def("getNumberofSM", &getNumberofSM);
  
  //1st try
  m.def("update_images", &update_images);

  //2nd try
  m.def("copy_parted_image_to_device", &copy_parted_image_to_device);
  m.def("update_images_v2", &update_images_v2);
  m.def("allocate_device", &allocate_device);

  //3rd try
  m.def("update_images_stream", &update_images_stream);

  //4th try, 
  //first part still return a float, not an array
  // m.def("copy_to_device", &copy_to_device, py::return_value_policy::copy);
  m.def("copy_to_device", [](size_t gpu_image, py::array_t<double, py::array::c_style> image, int size)
    {
      py::buffer_info bufImg = image.request();

      double *host_image = static_cast<double*>(bufImg.ptr);
      double *device_image = reinterpret_cast<double*>(gpu_image);

      hipMemcpy(device_image, host_image, size * sizeof(double), hipMemcpyHostToDevice);
    } 
  );
  
  //4th try, second part, somehow without any return, it is working
  m.def("update_images_v4", [](size_t gpu_image, size_t gpu_partial_update, double update, int size) 
    {
      //get the value of the address
      double *device_image = reinterpret_cast<double*>(gpu_image);
      double *device_partial_update = reinterpret_cast<double*>(gpu_partial_update);

      int devId, numSMs;
      hipGetDevice(&devId);
      hipDeviceGetAttribute( &numSMs, hipDeviceAttributeMultiprocessorCount, devId);
      
      partial_image_update<<<8*numSMs, 256>>>(device_image, device_partial_update, update, size);

      hipDeviceSynchronize();
    }
  );
}



