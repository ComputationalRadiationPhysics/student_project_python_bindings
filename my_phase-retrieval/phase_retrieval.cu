#include <stdio.h>
#include <string>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

//store C++ and CUDA version of phase retrieval
#include "phase_algo.hpp"
#include "test_algo.hpp"

namespace py = pybind11;

PYBIND11_MODULE(cuPhaseRet, m) 
{
  m.def("fienup_phase_retrieval", &fienup_phase_retrieval);
  m.def("array_check", &array_check);
  m.def("array_check_cuda", &array_check_cuda);
  m.def("array_check_complex", &array_check_complex);
  m.def("array_check_complex_cuda", &array_check_complex_cuda);
}


