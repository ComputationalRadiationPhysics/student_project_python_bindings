#include <stdio.h>
#include <string>
#include <hipfft/hipfft.h>
#include <pybind11/pybind11.h>
#include <pybind11/numpy.h>

//store C++ and CUDA version of phase retrieval
#include "phase_algo.hpp"

namespace py = pybind11;

PYBIND11_MODULE(cuPhaseRet, m) 
{
  //main phase retrieval
  m.def("fienup_phase_retrieval", py::overload_cast<py::array_t<double, py::array::c_style>, py::array_t<double, py::array::c_style>, int, string, double, py::array_t<double, py::array::c_style>>(&fienup_phase_retrieval));
  m.def("fienup_phase_retrieval", py::overload_cast<py::array_t<double, py::array::c_style>, py::array_t<double, py::array::c_style>, int, string, double>(&fienup_phase_retrieval));
}


